#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include "lib/libbmp.h"


__global__ void Kernel01(int size_filter, double *FilterMatrix, unsigned char* matrix_orig, int height, int width, unsigned char* matrix_filt) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int i = row;
    int j = col;
    if (row < height && col < width) {
        //bmp_img img_filt;
        //bmp_img_init_df(&img_filt, height, width);
        //for (size_t i = 0; i < height; i++) { // image row
        //    for (size_t j = 0; j < width; j++) { // pixels in image row
        float accumulator_red = 0;
        float accumulator_green = 0;
        float accumulator_blue = 0;
        //int count = 0;
        // position mask:
        //def f(i,j,k,l):
        //    return (i+k-1,j+l-1)
        for (size_t k = 0; k < size_filter; k++) { // kernel rows
            for (size_t l = 0; l < size_filter; l++) { // kernel elements/cols
                //if ((i % size_filter == k) && (j % size_filter == l)) {// corresponding element
                accumulator_red += FilterMatrix[k*size_filter + l] * (unsigned int) (matrix_orig[((i+k-1)*width + (j+l-1))*size_filter + 0]);
                accumulator_green += FilterMatrix[k*size_filter + l] * (unsigned int) (matrix_orig[((i+k-1)*width + (j+l-1))*size_filter + 1]);
                accumulator_blue += FilterMatrix[k*size_filter + l] * (unsigned int) (matrix_orig[((i+k-1)*width + (j+l-1))*size_filter + 2]);
                //printf("%d %f %f %d \n", (unsigned int) accumulator_red, accumulator_red, FilterMatrix[k*size_filter + l], (int) (matrix_orig[(i*width + j)*size_filter + 0]));
                //count += 1;
                //}
            }
        }
        //printf("%d\n", count);
        matrix_filt[(i*width + j)*size_filter + 0]= (unsigned int) accumulator_red;
        matrix_filt[(i*width + j)*size_filter + 1] = (unsigned int) accumulator_green;
        matrix_filt[(i*width + j)*size_filter + 2] = (unsigned int) accumulator_blue;
        //   }
        //}
        //return img_filt;
    }
}




void img_to_matrix(bmp_img img, int height, int width, unsigned char* matrix) {
    for (size_t i = 0; i < height; i++) { // image row
        for (size_t j = 0; j < width; j++) { // pixels in image row
            matrix[(i*width +j)*3 + 0] = img.img_pixels[i][j].red;
            matrix[(i*width +j)*3 + 1] = img.img_pixels[i][j].green;
            matrix[(i*width +j)*3 + 2] = img.img_pixels[i][j].blue;
        }


    }
}

void matrix_to_img(unsigned char* matrix, int height, int width, bmp_img img) {
    for (size_t i = 0; i < height; i++) { // image row
        for (size_t j = 0; j < width; j++) { // pixels in image row
            img.img_pixels[i][j].red = matrix[(i*width +j)*3 + 0];
            img.img_pixels[i][j].green = matrix[(i*width +j)*3 + 1];
            img.img_pixels[i][j].blue = matrix[(i*width +j)*3 + 2];
        }
    }
}



void seq(int size_filter, double *FilterMatrix, unsigned char* matrix_orig, int height, int width, unsigned char* matrix_filt) {
    //int row = blockIdx.y * blockDim.y + threadIdx.y;
    //int col = blockIdx.x * blockDim.x + threadIdx.x;
    //int i = row;
    //int j = col;
    //if (row < height && col < width) {
    //bmp_img img_filt;
    //bmp_img_init_df(&img_filt, height, width);
    for (size_t i = size_filter/2; i < height-size_filter/2; i++) { // image row
        for (size_t j = size_filter/2; j < width-size_filter/2; j++) { // pixels in image row
            float accumulator_red = 0;
            float accumulator_green = 0;
            float accumulator_blue = 0;
            //int count = 0;
            // position mask:
            //def f(i,j,k,l):
            //    return (i+k-1,j+l-1)
            for (size_t k = 0; k < size_filter; k++) { // kernel rows
                for (size_t l = 0; l < size_filter; l++) { // kernel elements/cols
                    //if ((i % size_filter == k) && (j % size_filter == l)) {// corresponding element
                    accumulator_red += FilterMatrix[k*size_filter + l] * (unsigned int) (matrix_orig[((i+k-1)*width + (j+l-1))*size_filter + 0]);
                    accumulator_green += FilterMatrix[k*size_filter + l] * (unsigned int) (matrix_orig[((i+k-1)*width + (j+l-1))*size_filter + 1]);
                    accumulator_blue += FilterMatrix[k*size_filter + l] * (unsigned int) (matrix_orig[((i+k-1)*width + (j+l-1))*size_filter + 2]);
                    //printf("%d %f %f %d \n", (unsigned int) accumulator_red, accumulator_red, FilterMatrix[k*size_filter + l], (int) (matrix_orig[(i*width + j)*size_filter + 0]));
                    //count += 1;
                    //}
                }
            }
            //printf("%d\n", count);
            matrix_filt[(i*width + j)*size_filter + 0]= (unsigned int) accumulator_red;
            matrix_filt[(i*width + j)*size_filter + 1] = (unsigned int) accumulator_green;
            matrix_filt[(i*width + j)*size_filter + 2] = (unsigned int) accumulator_blue;

        }
    }
    //return img_filt;
    //}
}


int main (int argc, char *argv[])
{
    if (argc!=2) {
        printf("Usage: ./cuda image_to_be_filter\n");
        exit(1);
    }

    bmp_img h_img_orig;
    bmp_img_read(&h_img_orig, argv[1]);
    int width = (int) h_img_orig.img_header.biWidth;
    int height = (int) h_img_orig.img_header.biHeight;
    int channels = 3;
    printf("La imagen es %d X %d\n", width, height);

    unsigned char *h_matrix_orig;
    h_matrix_orig = (unsigned char *) malloc (sizeof(unsigned char)*height*width*channels);

    img_to_matrix(h_img_orig, height, width, h_matrix_orig);


    unsigned char *matrix_filt;
    matrix_filt = (unsigned char *) malloc (sizeof(unsigned char)*height*width*channels);
    double h_K[9] = {0.11,0.11,0.11,0.11,0.11,0.11,0.11,0.11,0.11};

    // Sequential
    seq(channels, h_K, h_matrix_orig, width, height, matrix_filt);


    // Cuda 1
    unsigned char *d_matrix_orig, *d_matrix_filt, *h_matrix_filt;
    h_matrix_filt = (unsigned char *) malloc (sizeof(unsigned char)*height*width*channels);
    double *d_K;

    int numBytesK = sizeof(h_K);
    int numBytesOrig = sizeof(char)*height*width*channels;
    int numBytesFilt = sizeof(char)*height*width*channels;


    // Obtener Memoria en el device
    hipMalloc((double**)&d_K, numBytesK);
    hipMalloc((unsigned char ***)&d_matrix_orig, numBytesOrig);
    hipMalloc((unsigned char ***)&d_matrix_filt, numBytesFilt);

    // Copiar datos desde el host en el device
    hipError_t err = hipMemcpy(d_K, h_K, numBytesK, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("1: CUDA error copying to Device: %s\n", hipGetErrorString(err));
    }
    err = hipMemcpy(d_matrix_orig, h_matrix_orig, numBytesOrig, hipMemcpyHostToDevice);
    if (err!=hipSuccess) {
        printf("2: CUDA error copying to Device: %s\n", hipGetErrorString(err));
    }

    // Ejecutar el kernel
    int SIZE = 32;
    int nThreads = SIZE;
    int N = width;
    int M = height;

    // numero de Blocks en cada dimension
    int nBlocksN = (N+nThreads-1)/nThreads;
    int nBlocksM = (M+nThreads-1)/nThreads;
    dim3 dimGrid(nBlocksM, nBlocksN, 1);
    dim3 dimBlock(nThreads, nThreads, 1);
    Kernel01<<<dimGrid, dimBlock>>>(channels, d_K, d_matrix_orig, width, height, d_matrix_filt);

    if (hipSuccess != hipGetLastError())
        printf("3: CUDA error at kernel exec: %s\n", hipGetErrorString(hipGetLastError()));
    err = hipMemcpy(h_matrix_filt, d_matrix_filt, numBytesFilt, hipMemcpyDeviceToHost);
    if (err!=hipSuccess) {
        printf("4: CUDA error copying to Host: %s\n", hipGetErrorString(err));
    }

    hipFree(d_K);
    hipFree(d_matrix_orig);
    hipFree(d_matrix_filt);


    //Test
    bmp_img img_filt;
    bmp_img_init_df(&img_filt, height, width);
    matrix_to_img(matrix_filt, height, width, img_filt);
    bmp_img_write(&img_filt, strcat("SEQ", argv[1]));

    bmp_img h_img_filt;
    bmp_img_init_df(&h_img_filt, height, width);
    matrix_to_img(h_matrix_filt, height, width, h_img_filt);
    bmp_img_write(&h_img_filt, strcat("CUDA1", argv[1]));

    free(h_matrix_orig);
    free(h_matrix_filt);
    free(matrix_filt);


    bmp_img_free(&h_img_orig);
    bmp_img_free(&h_img_filt);
    bmp_img_free(&img_filt);




    return 0;

}
